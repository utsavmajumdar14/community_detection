#include <hip/hip_runtime.h>
#include <stdio.h>


#include <iostream>

using namespace std;

#define BLOCK_SIZE 16

__global__ void degreeKernel(float *d_adjacency_matrix, int *degrees, int nodes)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < nodes)
    {
        int degree = 0;
        for(int i = 0; i < nodes; i++) {
            degree += d_adjacency_matrix[tid * nodes + i] != 0;
        }
        degrees[tid] = degree;
    }
}

__global__ void common_neighbor_kernel(int nodes, float* adjacency_matrix, int* int_neigh_count)
{
    __shared__ float shared_matrix[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int shared_counts[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    shared_counts[ty][tx] = 0;

    if (row < nodes && col < nodes)
        shared_matrix[ty][tx] = adjacency_matrix[row * nodes + col];
    else
        shared_matrix[ty][tx] = 0;

    __syncthreads();

    if (row < nodes && col < nodes && row != col)
        for (int k = 0; k < BLOCK_SIZE; k++)
            if (shared_matrix[ty][k] != 0 && shared_matrix[k][tx] != 0 && shared_matrix[ty][tx] != 0)
                shared_counts[ty][tx]++;

    __syncthreads();

    if (row < nodes && col < nodes && row != col)
        atomicAdd(&int_neigh_count[row * nodes + col], shared_counts[ty][tx]);

}

__global__ void compute_neigh_count(float *d_adjacency_matrix, int *d_int_neigh_count, int *d_neigh_count, int nodes)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<nodes && j<nodes)
    {
        if(d_adjacency_matrix[i*nodes+j] > 0)
            d_neigh_count[i*nodes+j] = d_int_neigh_count[i*nodes+j] + 1;
        else
            d_neigh_count[i*nodes+j] = d_int_neigh_count[i*nodes+j];
    }
}

__global__ void compute_wtd_neigh_count(int *d_neigh_count, int *d_degrees, float *d_wtd_neigh_count, int nodes)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<nodes && j<nodes)
    {
        int index = i * nodes + j;
        d_wtd_neigh_count[index] = d_neigh_count[index]*1.0/d_degrees[i];
    }

}

__global__ void add_prod_kernel(float *d_adjacency_matrix, float *d_wtd_neigh_count, float *d_added_neigh_count, float *d_prod_neigh_count, int nodes)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i<nodes && j<nodes)
    {
        int index = i * nodes + j;
        d_added_neigh_count[index] = d_adjacency_matrix[index] + d_wtd_neigh_count[index];
        d_prod_neigh_count[index] = d_adjacency_matrix[index] * d_wtd_neigh_count[index];
    }
}


int main()
{
    const int nodes = 10;
    const int adjacency_size = nodes * nodes;


    float h_adjacency_matrix[nodes][nodes] = {
        {0, 1.0, 0.7, 0, 0, 0, 0, 0, 0, 0},
        {1.0, 0, 0.8, 0, 0, 0, 0.2, 0, 0, 0},
        {0.7, 0.8, 0, 0.3, 0, 0, 0, 0, 0, 0},
        {0, 0, 0.3, 0, 0.7, 0.8, 0, 0, 0, 0},
        {0, 0, 0, 0.7, 0, 0.9, 0, 0, 0, 0},
        {0, 0, 0, 0.8, 0.9, 0, 0, 0, 0, 0.3},
        {0, 0.2, 0, 0, 0, 0, 0, 0.9, 0.7, 0.8},
        {0, 0, 0, 0, 0, 0, 0.9, 0, 0.5, 0.6},
        {0, 0, 0, 0, 0, 0, 0.7, 0.5, 0, 0.4},
        {0, 0, 0, 0, 0, 0.3, 0.8, 0.6, 0.4, 0}
    };

    float* d_adjacency_matrix;
    int* d_degrees;
    int* d_int_neigh_count;
    int *d_neigh_count;
    float *d_wtd_neigh_count;
    float *d_added_neigh_count;
    float *d_prod_neigh_count;



    hipMalloc((void **)&d_adjacency_matrix, adjacency_size * sizeof(float));
    hipMalloc((void **)&d_degrees, nodes * sizeof(int));
    hipMalloc((void **)&d_int_neigh_count, adjacency_size * sizeof(int));
    hipMalloc((void **)&d_neigh_count, adjacency_size * sizeof(int));
    hipMalloc((void **)&d_wtd_neigh_count, adjacency_size*sizeof(float));
    hipMalloc((void **)&d_added_neigh_count, adjacency_size*sizeof(float));
    hipMalloc((void **)&d_prod_neigh_count, adjacency_size*sizeof(float));

    hipMemset(d_int_neigh_count, 0, adjacency_size * sizeof(int));
    hipMemset(d_neigh_count, 0, adjacency_size*sizeof(int));
    hipMemset(d_wtd_neigh_count, 0.0, adjacency_size*sizeof(float));
    hipMemset(d_added_neigh_count, 0.0, adjacency_size*sizeof(float));
    hipMemset(d_prod_neigh_count, 0.0, adjacency_size*sizeof(float));

    hipMemcpy(d_adjacency_matrix, h_adjacency_matrix, adjacency_size * sizeof(float), hipMemcpyHostToDevice);

    //Launch Degree Kernel
    degreeKernel<<<(nodes + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_adjacency_matrix, d_degrees, nodes);

    int h_degrees[nodes];
    hipMemcpy(h_degrees, d_degrees, nodes * sizeof(int), hipMemcpyDeviceToHost);


    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((nodes + BLOCK_SIZE - 1) / BLOCK_SIZE, (nodes + BLOCK_SIZE - 1) / BLOCK_SIZE);

    common_neighbor_kernel<<<grid_size, block_size>>>(nodes, d_adjacency_matrix, d_int_neigh_count);

    hipDeviceSynchronize();

    int h_int_neigh_count[nodes][nodes];
    hipMemcpy(h_int_neigh_count, d_int_neigh_count, adjacency_size * sizeof(int), hipMemcpyDeviceToHost);

    cout<<"\n Intermediate Neighbourhood Count : \n";
    for(int i=0; i<nodes; i++)
    {
        for(int j=0; j<nodes; j++)
            cout << h_int_neigh_count[i][j] << " ";
        cout << endl;
    }

    compute_neigh_count<<<grid_size, block_size>>>(d_adjacency_matrix, d_int_neigh_count, d_neigh_count, nodes);

    hipDeviceSynchronize();

    int h_neigh_count[nodes][nodes];
    hipMemcpy(h_neigh_count, d_neigh_count, adjacency_size*sizeof(int), hipMemcpyDeviceToHost);

    cout<<"\n Neighbourhood Count : \n";
    for(int i=0; i<nodes; i++)
    {
        for(int j=0; j<nodes; j++)
            cout << h_neigh_count[i][j] << " ";
        cout << endl;
    }

    compute_wtd_neigh_count<<<grid_size, block_size>>>(d_neigh_count, d_degrees, d_wtd_neigh_count, nodes);

    hipDeviceSynchronize();

    float h_wtd_neigh_count[nodes][nodes];
    hipMemcpy(h_wtd_neigh_count, d_wtd_neigh_count, adjacency_size*sizeof(float), hipMemcpyDeviceToHost);

    for(int i=0; i<nodes; i++)
        cout<<h_degrees[i]<<"\n";

    cout<<"\n Weighted Neighbourhood Count : \n";
    for(int i=0; i<nodes; i++)
    {
        for(int j=0; j<nodes; j++)
            cout << h_wtd_neigh_count[i][j] << " ";
        cout << endl;
    }

    add_prod_kernel<<<grid_size, block_size>>>(d_adjacency_matrix, d_wtd_neigh_count, d_added_neigh_count, d_prod_neigh_count, nodes);

    hipDeviceSynchronize();

    float h_added_neigh_count[nodes][nodes];
    float h_prod_neigh_count[nodes][nodes];

    hipMemcpy(h_added_neigh_count, d_added_neigh_count, adjacency_size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_prod_neigh_count, d_prod_neigh_count, adjacency_size*sizeof(float), hipMemcpyDeviceToHost);

    cout<<"\n Added Neighbourhood Count : \n";
    for(int i=0; i<nodes; i++)
    {
        for(int j=0; j<nodes; j++)
            cout << h_added_neigh_count[i][j] << " ";
        cout << endl;
    }

    cout<<"\n Product Neighbourhood Count : \n";
    for(int i=0; i<nodes; i++)
    {
        for(int j=0; j<nodes; j++)
            cout << h_prod_neigh_count[i][j] << " ";
        cout << endl;
    }

    hipFree(d_degrees);
    hipFree(d_adjacency_matrix);
    hipFree(d_int_neigh_count);
    hipFree(d_neigh_count);
    hipFree(d_wtd_neigh_count);
    hipFree(d_added_neigh_count);
    hipFree(d_prod_neigh_count);

}
