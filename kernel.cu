#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <unordered_map>
#include <random>
#include <cmath>

using namespace std;


__global__ void vecAdd(float* a, float* b, float* c, float* d, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) 
    {
        if (a[i] == b[i])
        { 
            if (b[i] == c[i])
            {
                d[i] = 0;
            }
            else
            { 
                d[i] = 1;
            }
        }
        else
        { 
            d[i] = 1;
        }
    }
}

int main()
{
    const int N = 100;
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* a, * b, * c, *d;
    float* d_a, * d_b, * d_c, *d_d;

    // Allocate memory on the host
    a = (float*)malloc(N * sizeof(float));
    b = (float*)malloc(N * sizeof(float));
    c = (float*)malloc(N * sizeof(float));
    d = (float*)malloc(N * sizeof(float));

    // Allocate memory on the device
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));
    hipMalloc(&d_d, N * sizeof(float));

    // Initialize arrays a and b
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
        c[i] = i;
    }

    // Copy arrays a and b to the device
    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N * sizeof(float), hipMemcpyHostToDevice);

    // Perform vector addition on the device
    vecAdd << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_c, d_d,N);

    // Copy the result back to the host
    hipMemcpy(d, d_d, N * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%f\t", d[i]);
    }


    /*
    int flag = 1;
    // Verify the result
    for (int i = 0; i < N; i++) {
        if (c[i] != a[i] + b[i]) {
            printf("Error: c[%d] = %f, expected %f\n", i, c[i], a[i] + b[i]);
            flag = 0;
            break;
        }
    }
    printf("HELLO\n");
    printf("%f\n",c[0]);
    printf("%f\n", c[77]);
    printf("%f\n", c[9999999]);
    printf("%d", flag);
    */


    // Free memory
    free(a);
    free(b);
    free(c);
    free(d);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);

    return 0;
}
